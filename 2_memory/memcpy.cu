
#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
  // 1. Set host memory
  const int SIZE = 8;
  const int BYTES = SIZE * sizeof(float);
  const float a[SIZE] = {1.f, 2.f, 3.f, 4.f, 5.f, 6.f, 7.f, 8.f};
  float b[SIZE] = {0.f,};

  // 2. Set pointer
  float *dev_a = nullptr;
  float *dev_b = nullptr;

  // 3. Allocate device memory
  if (hipMalloc((void**)&dev_a, BYTES) != hipSuccess) {
    printf("hipMalloc dev_a failed!\n");
    return 1;
  }
  if (hipMalloc((void**)&dev_b, BYTES) != hipSuccess) {
    printf("hipMalloc dev_b failed!\n");
    return 1;
  }

  // 4. Copy host memory to device
  if (hipMemcpy(dev_a, a, BYTES, hipMemcpyHostToDevice) != hipSuccess) { // dev_a = a
    printf("hipMemcpy a to dev_a failed!\n");
    return 1;
  }
  if (hipMemcpy(dev_b, dev_a, BYTES, hipMemcpyDeviceToDevice) != hipSuccess) {// dev_b = dev_a
    printf("hipMemcpy dev_a to dev_b failed!\n");
    return 1;
  }
  if (hipMemcpy(b, dev_b, BYTES, hipMemcpyDeviceToHost) != hipSuccess) { // b = dev_b
    printf("hipMemcpy dev_b to b failed!\n");
    return 1;
  }

#if defined(__linux__)
  hipDeviceSynchronize();
#endif

  // 5. Check result
  for (int i = 0; i < SIZE; ++i)
  {
    printf("b[%d]: %.2f\n", i, b[i]);
  }

  // 6. Free device memory
  if (hipFree(dev_a) != hipSuccess) {
    printf("hipFree dev_a failed!\n");
  }
  if (hipFree(dev_b) != hipSuccess) {
    printf("hipFree dev_b failed!\n");
  }

  return 0;
}