#include <iostream>
#include <chrono>
#include <cstdlib>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


template<typename PREC>
void setRandomData(PREC data[], const int size, const PREC bound=static_cast<PREC>(1000)) {
  int bnd = static_cast<int>(bound);
  for (int i = 0; i < size; i++) {
    data[i] = (rand() % bnd) / bound;
  }
}

template<typename PREC>
PREC getSum(const PREC data[], const int size) {
  PREC sum = static_cast<PREC>(0);
  for (int i = 0; i < size; i++) {
    sum += data[i];
  }
  return sum;
}

template<typename PREC>
__global__ void kernelVecAddGPU(PREC vecC[], const PREC vecA[], const PREC vecB[], const int size) {
  unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < size) {
    vecC[id] = vecA[id] + vecB[id];
  }
}

int main()
{
  const int SIZE = 256 * 1024 * 1024;
  float *vecA = new float[SIZE];
  float *vecB = new float[SIZE];
  float *vecC = new float[SIZE];
  std::chrono::steady_clock::time_point begin;
  std::chrono::steady_clock::time_point end;
  std::chrono::microseconds elapsed_time;

  srand(0);
  setRandomData(vecA, SIZE);
  setRandomData(vecB, SIZE);

  float *d_vecA, *d_vecB, *d_vecC;
  hipMalloc((void**)&d_vecA, SIZE * sizeof(float));
  hipMalloc((void**)&d_vecB, SIZE * sizeof(float));
  hipMalloc((void**)&d_vecC, SIZE * sizeof(float));
  begin = std::chrono::steady_clock::now();
  hipMemcpy(d_vecA, vecA, SIZE * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_vecB, vecB, SIZE * sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
  elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>(end - begin);
  std::cout << "Data transfer to GPU Elapsed time: " << static_cast<float>(elapsed_time.count()) / 1000 << "ms" << std::endl;

  dim3 dimBlock(1024, 1, 1);
  dim3 dimGrid((SIZE + dimBlock.x - 1) / dimBlock.x, 1, 1);
  begin = std::chrono::steady_clock::now();
  kernelVecAddGPU<<<dimGrid, dimBlock>>>(d_vecC, d_vecA, d_vecB, SIZE);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("kernel launch failed: %s\n", hipGetErrorString(err));
    return 1;
  }
  end = std::chrono::steady_clock::now();
  elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>(end - begin);
  std::cout << "GPU Elapsed time: " << static_cast<float>(elapsed_time.count()) / 1000 << "ms" << std::endl;

  hipMemcpy(vecC, d_vecC, SIZE * sizeof(float), hipMemcpyDeviceToHost);
  float sumA = getSum(vecA, SIZE);
  float sumB = getSum(vecB, SIZE);
  float sumC = getSum(vecC, SIZE);
  float sumDiff = fabsf(sumC - (sumA + sumB));

  std::cout << "Sum A: " << sumA << std::endl;
  std::cout << "Sum B: " << sumB << std::endl;
  std::cout << "Sum C: " << sumC << std::endl;
  std::cout << "Sum Difference: " << sumDiff << std::endl;

  delete[] vecA;
  delete[] vecB;
  delete[] vecC;
  hipFree(d_vecA);
  hipFree(d_vecB);
  hipFree(d_vecC);

  return 0;
}