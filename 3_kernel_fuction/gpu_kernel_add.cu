#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


__global__ void add_kernel(int *c, const int *a, const int *b) {
  int idx = threadIdx.x;
  c[idx] = a[idx] + b[idx];
}

int main()
{
  // 1. Set host memory
  const int SIZE = 5;
  const int BYTES = SIZE * sizeof(int);
  const int a[SIZE] = {1, 2, 3, 4, 5};
  const int b[SIZE] = {10, 20, 30, 40, 50};
  int c[SIZE] = {0,};

  // 2. Set Pointers for device memory
  int* dev_a = nullptr;
  int* dev_b = nullptr;
  int* dev_c = nullptr;


  // 3. Allocate device memory
  if (hipMalloc((void**)&dev_a, BYTES) != hipSuccess) {
    printf("hipMalloc dev_a failed\n");
    return 1;
  }
  if (hipMalloc((void**)&dev_b, BYTES) != hipSuccess) {
    printf("hipMalloc dev_b failed\n");
    return 1;
  }
  if (hipMalloc((void**)&dev_c, BYTES) != hipSuccess) {
    printf("hipMalloc dev_c failed\n");
    return 1;
  }

  // 4. Copy source host memory to device memory
  if (hipMemcpy(dev_a, a, BYTES, hipMemcpyHostToDevice) != hipSuccess) {
    printf("hipMemcpy host to device failed\n");
    return 1;
  }
  if (hipMemcpy(dev_b, b, BYTES, hipMemcpyHostToDevice) != hipSuccess) {
    printf("hipMemcpy host to device failed\n");
    return 1;
  }

  // 5. Launch kernel & Check error 
  add_kernel<<<1, SIZE>>>(dev_c, dev_a, dev_b);
#if defined(__linux__)
  hipDeviceSynchronize();
#endif
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("kernel launch failed: %s\n", hipGetErrorString(err));
    return 1;
  } else {
    printf("kernel launch success!\n");
  }

  //6. Copy result device memory to host memory
  if (hipMemcpy(c, dev_c, BYTES, hipMemcpyDeviceToHost) != hipSuccess) {
    printf("hipMemcpy device to host failed\n");
    return 1;
  }

  // 7. Free device memory
  if (hipFree(dev_a) != hipSuccess) {
    printf("hipFree dev_a failed\n");
    return 1;
  }
  if (hipFree(dev_b) != hipSuccess) {
    printf("hipFree dev_b failed\n");
    return 1;
  }
  if (hipFree(dev_c) != hipSuccess) {
    printf("hipFree dev_c failed\n");
    return 1;
  }

  // 8. Print result
  printf("c[i] = a[i] + b[i]\n");
  printf("%d %d %d %d %d\n", c[0], c[1], c[2], c[3], c[4]);

  return 0;
}