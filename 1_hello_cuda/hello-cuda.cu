#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


__global__ void hello() {
  printf("Hello CUDA! %d\n", threadIdx.x);
}

int main() {
  hello<<<2, 5>>>();

#if defined(__linux__)
  hipDeviceSynchronize();
#endif

  return 0;
}